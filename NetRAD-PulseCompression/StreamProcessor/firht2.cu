//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%% Module: Stream Processing Techniques for OptiSDR %%%%%%%%%%%%%%
//%%%%%%%%%%%%%% Author: Lerato J. Mohapi %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%% Institute: University of Cape Town %%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%% Inlcude some C Libraries %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cstdlib>
#include <sys/time.h>
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%%% Include Cuda run-time and inline Libraries %%%%%%%%%%%%%%%%%
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

using namespace std;
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%% Including the Kernel: FIR Filter Calculator %%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%% NetRAD Raw ADC Data Read %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
void readNetRadSamples(string strFilename, unsigned int uiNSamples, vector<short> &vsSamples)
{
	//Read
	ifstream oIFS;
	oIFS.open(strFilename.c_str(), ifstream::binary);
	if(!oIFS.is_open())
	{	
		cout << "[SDR_DSL_INFO]$ Error unable to open file \"" << strFilename << "\"" << endl;
		oIFS.close();
		exit(1);
	}

	vsSamples.resize(uiNSamples);

	oIFS.read((char*)&vsSamples.front(), sizeof(short) * uiNSamples);

	if(oIFS.gcount() << sizeof(short) * uiNSamples && oIFS.eof())
	{
		cout << "[SDR_DSL_INFO]$ Warning: hit end of file after " << oIFS.gcount() / sizeof(short) << " samples. Output is shortened accordingly." << endl;
		vsSamples.resize(oIFS.gcount() / sizeof(short));
	}

	oIFS.close();

	//int iTemp;

	//Covert from unsigned to signed - Uncomment lines below
	/*for(unsigned int uiSampleNo = 0; uiSampleNo < uiNSamples; uiSampleNo++)
	{
		iTemp = *((unsigned short*)&vsSamples[uiSampleNo]);
		iTemp -= 8192;
		vsSamples[uiSampleNo] = iTemp;
	}*/
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%% Complex Multiplication Kernel %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__global__ void complexvector_multiply(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < N)
		outp[tid] = hipCmulf(_cx[tid],_cy[tid]);
}
//
// This Kernel Utilize a 1-D Grid to a 1-D data indexes (1-D BlockDim = 1024 threads)
// We can multiply 16 Million floating point values in Parallel
__global__ void complexvector_multiply1d1d(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	//
	if(index < N)
	{
		outp[index].x = (_cx[index].x*_cy[index].x) - (_cx[index].y*_cy[index].y);//cuCmulf(_cx[index],_cy[index]);
		outp[index].y = (_cx[index].x*_cy[index].y) + (_cx[index].y*_cy[index].x);
	}
}
// This Kernel Utilize a 2-D Grid flattened to a 1-D data indexes (1-D BlockDim = 1024 threads)
// We can multiply 68.747788288 Million floating point values in Parallel
__global__ void complexvector_multiply2d1d(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
	//
	int xsize = blockDim.x*gridDim.x; // X dimension total number of thread
	//
	int index = tidy*xsize + tidx; // Index through 2-D grid
	//
	if(index < N)
		outp[index] = hipCmulf(_cx[index],_cy[index]);
}
// This Kernel Utilize a 3-D Grid flattened to a 1-D data indexes
// We can multiply 129 Million floating point values in Parallel
__global__ void complexvector_multiply3d1d(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
	int tidz = threadIdx.z + blockIdx.z * blockDim.z;
	//
	int xsize = blockDim.x*gridDim.x; // X dimension total number of thread
	int zsize = xsize*blockDim.y*gridDim.y; // Entire 2-D grid numer of thread
	//
	int xyindex = tidy*xsize + tidx; // Index through 2-D grid
	int index = tidz*zsize + xyindex; // Index through entire 3-D grid
	//
	if(index < N)
		outp[index] = hipCmulf(_cx[index],_cy[index]);
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%% Complex Conjugate Kernel %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__global__ void complexvector_conjugate(hipFloatComplex *_cx, hipFloatComplex *outp,int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		//outp[tid].x= (_cx[tid].x*_cy[tid].x)-(_cx[tid].y*_cy[tid].y);
		//outp[tid].y= (_cx[tid].x*_cy[tid].y)+(_cy[tid].x*_cx[tid].y);
		outp[tid] = hipConjf(_cx[tid]);
		tid += blockDim.x * gridDim.x;
	}
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%% Complex Absolute Kernel %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__global__ void complexvector_abs(hipFloatComplex *_cx,float *outp,int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		//outp[tid].x= (_cx[tid].x*_cy[tid].x)-(_cx[tid].y*_cy[tid].y);
		//outp[tid].y= (_cx[tid].x*_cy[tid].y)+(_cy[tid].x*_cx[tid].y);
		outp[tid] = hipCabsf(_cx[tid]);
		tid += blockDim.x * gridDim.x;
	}
}
//
__global__ void optisdrifftscale(hipFloatComplex *invec, hipFloatComplex *out, int fp, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < N)
		out[tid] = make_hipFloatComplex(hipCrealf(invec[tid])/(float)fp,hipCimagf(invec[tid])/(float)fp);
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%% Some Function Prototypes %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
void getShortSamples(vector<short> &vSamples, unsigned int nSamples);
void getRealSamples(vector<float> &vSamples, unsigned int nSamples);
float* getRealFSamples(int dalen);
void getRandSamples(vector<float> &vSamples, unsigned int nSamples);
void getAscendingRandSamples(vector<float> &vSamples, unsigned int nSamples);
hipFloatComplex* getComplexSamples(int _dlen);
hipFloatComplex* getComplexRandomSamples(int _dlen);

hipFloatComplex* getComplexSamples(vector<short> &vSamples, int _dlen)
{
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	for(int i = 0; i< _dlen; i++)
	{
		cSamples[i] = make_hipFloatComplex(vSamples[i],0.0);
	}
	return cSamples;
}
//
hipFloatComplex* getComplexSamples(vector<short> &vSamples, int from, int to)
{
	int dlen = to-from;
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(dlen*sizeof(hipFloatComplex));
	for(int i = from; i< to; i++)
	{
		cSamples[i%dlen] = make_hipFloatComplex(vSamples[i],0.0);
	}
	return cSamples;
}
//
//
hipFloatComplex* getComplexEmpty(int _dlen)
{
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	/*for(int i = 0; i< _dlen; i++)
	{
		cSamples[i] = make_cuFloatComplex(0.0,0.0);
	}*/
	return cSamples;
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%% Error Checker forv CUDA Function Calls %%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
#define cuErrorCheck(ans) { cuErrorChecker((ans), __FILE__, __LINE__); }
inline void cuErrorChecker(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%% Stream Processing using CUDA GPUs %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void streamprocessor(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> hout, int dsize,int chunk)
{
	//
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	//hdata.resize(dsize); // This must be done outside with data initialization/generation
	//hout.resize(dsize); // Might be a good idea to initialize outside of this function...
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);	
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	// Trying to Create Page-Locked std::vector - No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);//cudaHostAlloc(...);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	
	for(int i = 0; i<dsize; i++)
	{
		hipfftPlan1d(&plans[i],chunk,HIPFFT_C2C,1);
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
    		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		//free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	gettimeofday(&t2, 0);
	double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	double time2 = ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000000.0;
	printf("\n[SDR_DSL_INFO]$ Exec. Time for StreamProcessed FFT = %f s ~= %f...!\n", time,time2);
	//cudaDeviceReset();
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%% Implementing a New Stream Processor Technique that Utilizes Batched CUFFT, this must increase Performance %%%
//%%% Reduce the number of streams by Performing Optimized Batched CUFFT, this must be Faster...Target is NetRAD %%
//%%% Data Size = chunk*dsize, where dsize = vector size (<= 100 preferably) %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%% ftpoint = FFT point, We know for sure that Batched CUFFT is performing well (2048x8192 ~= 0.1xx seconds) %%%%
//%%%%*********************************************************************************************************%%%%
#define N_SIGS  1300//
#define SIG_LEN 2048 //
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void streamprocessor(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	//struct timeval t1, t2;
	//gettimeofday(&t1, 0);
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1, SIG_LEN, //advanced data layout, NULL shuts it off
    		NULL, 1, SIG_LEN, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C, N_SIGS);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//cudaStreamCreate(&optisdr_streams[i]);
		//cudaMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(cuFloatComplex),cudaMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//cufftExecC2C(plans[i],ddata[i],dout[i],CUFFT_FORWARD);
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		//free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	//gettimeofday(&t2, 0);
	//double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	//double time2 = ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000000.0;
	//printf("\n[SDR_DSL_INFO]$ Exec. Time for StreamProcessed FFT = %f s ~= %f...!\n", time,time2);
	//cudaDeviceReset();
}
//

void readNetRadSamples2(string strFilename, unsigned int nsamples, vector<float> &vsSamples)
{
	//Read
	ifstream oIFS;
	oIFS.open(strFilename.c_str(),ifstream::in);
	if(!oIFS.is_open())
	{	
		cout << "[SDR_DSL_INFO]$ Error unable to open file \"" << strFilename << "\"" << endl;
		oIFS.close();
		exit(1);
	}
	//
	vsSamples.resize(nsamples);
	int i = 0;
	while(i < nsamples)
	{
		oIFS>>vsSamples[i];
		//vsSamples[i] = atof(tmp[i].c_str());
		//printf("%f, ",vsSamples[i]);
		i++;
	}
	oIFS.close();
}
//
hipFloatComplex* getReferenceSignal(int _dlen)
{
	hipFloatComplex *refsig = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	//
	vector<float> rSamples,iSamples;
	readNetRadSamples2("../data/rref3sig.dat",_dlen,rSamples);
	readNetRadSamples2("../data/iref3sig.dat",_dlen,iSamples);
	//
	for(int i = 0; i< _dlen; i++)
	{
		refsig[i] = make_hipFloatComplex(rSamples[i],iSamples[i]);
	}
	return refsig;
}
//
// This one accepts the return size and file size.
//
hipFloatComplex* getReferenceSignal(int fsize, int _dlen)
{
	hipFloatComplex *refsig = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	//
	//
	vector<float> rSamples,iSamples;
	readNetRadSamples2("../data/rref3sig.dat",fsize,rSamples);
	readNetRadSamples2("../data/iref3sig.dat",fsize,iSamples);
	//
	for(int i = 0; i< fsize; i++)
	{
		refsig[i] = make_hipFloatComplex(rSamples[i],iSamples[i]);
		// 
	}
	//
	for(int j = (_dlen-fsize); j< _dlen; j++)
	{
		refsig[j] = make_hipFloatComplex(0.0f,0.0f);
	}
	//
	return refsig;
}
//
hipFloatComplex* resizeVector(hipFloatComplex *inp, int oldlen, int newlen)
{
	hipFloatComplex *outp = (hipFloatComplex*)malloc(newlen*sizeof(hipFloatComplex));
	//
	for(int i = 0; i< newlen; i++)
	{
		outp[i] = inp[i%oldlen];
	}
	//
	//outp[0] = make_cuFloatComplex(2.0f,2.0f); // Just for testing: TODO: Remove when done
	//outp[2048] = make_cuFloatComplex(2.0f,2.0f); // Just for testing: TODO: Remove when done
	//outp[4096] = make_cuFloatComplex(2.0f,2.0f); // Just for testing: TODO: Remove when done
	return outp;
}

//
hipFloatComplex* getComplexSamples(vector<short> &vSamples, int from, int to, int chunksize, int outputsize)
{
	int dlen = outputsize*((to-from)/chunksize);
	int skip = outputsize-chunksize;
	//
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(dlen*sizeof(hipFloatComplex));
	//for(int i = 0;i<dlen;i++) cSamples[i] = make_cuFloatComplex(0.0f,0.0f);
	int idx = 0;
	if(skip>0)
	{
		for(int i = from; i< to; i++)
		{
		
			if(((i%chunksize)==0)&&(i>from))
			{
				idx+=skip; // We skip 
				cSamples[idx] = make_hipFloatComplex((float)vSamples[i],0.0f);
				idx+=1;
			}
			else
			{
				cSamples[idx] = make_hipFloatComplex((float)vSamples[i],0.0f);
				idx+=1;
			}
		}
	}
	return cSamples;
}
//
hipFloatComplex* getComplexEmpty(int _dlen, int chunksize, int outputsize)
{
	int dlen = (_dlen)*(outputsize/chunksize);
	//
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(dlen*sizeof(hipFloatComplex));
	//for(int i = 0;i<dlen;i++) cSamples[i] = make_cuFloatComplex(0.0f,0.0f);
	//
	return cSamples;
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void spMultiply(vector<hipFloatComplex*> hdata1, vector<hipFloatComplex*> hdata2,vector<hipFloatComplex*> ddata1, vector<hipFloatComplex*> ddata2, vector<hipFloatComplex*> dout, int spsize,int chunk)
{
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	hipStream_t optisdr_streams[spsize];
	//
	for(int i = 0;i<spsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	for(int i = 0;i<spsize;i++)
	{
		//
		hipMemcpyAsync(ddata1[i],hdata1[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		hipMemcpyAsync(ddata2[i],hdata2[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	// Execution
	for(int i = 0; i<spsize; i++)
	{		
		//
        	complexvector_multiply1d1d<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],ddata1[i],ddata2[i],chunk);
		hipStreamSynchronize(optisdr_streams[i]);
		//cudaMemcpyAsync(hout[i],dout[i],chunk*sizeof(cuFloatComplex),cudaMemcpyDeviceToHost,optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < spsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
	}
	//
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void hMultiply(vector<hipFloatComplex*> hdata, hipFloatComplex* refsig, vector<hipFloatComplex*> hout, int dsize,int chunk)
{
	//
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	//
	hipFloatComplex *drefsig,*dftrefsig;
	hipMalloc((void**)&drefsig,chunk*sizeof(hipFloatComplex));
	hipMalloc((void**)&dftrefsig,chunk*sizeof(hipFloatComplex));
	hipHostRegister(refsig,chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
	//
	vector<hipFloatComplex*> ddata,dout,drout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//
	}
	//
	hipMemcpyAsync(drefsig,refsig,chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[0]);
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		//
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		complexvector_multiply<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(ddata[i],dftrefsig,dout[i],chunk);
		// 
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		free(hdata[i]);
	}
	hipHostUnregister(refsig);
	hipFree(drefsig);
}
//
void dMultiply(vector<hipFloatComplex*> hdata, hipFloatComplex* refsig, vector<hipFloatComplex*> dout, int dsize,int chunk)
{
	//
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	//
	hipFloatComplex *drefsig,*dftrefsig;
	hipMalloc((void**)&drefsig,chunk*sizeof(hipFloatComplex));
	hipMalloc((void**)&dftrefsig,chunk*sizeof(hipFloatComplex));
	hipHostRegister(refsig,chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
	//
	vector<hipFloatComplex*> ddata;//,dout,drout;
	ddata.resize(dsize);
	//dout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//cudaHostRegister(hout[i],chunk*sizeof(cuFloatComplex),cudaHostRegisterPortable);
		//
	}
	//
	hipMemcpyAsync(drefsig,refsig,chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[0]);
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		//
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		complexvector_multiply<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(ddata[i],dftrefsig,dout[i],chunk);
		// 
		//cudaMemcpyAsync(hout[i],dout[i],chunk*sizeof(cuFloatComplex),cudaMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		//cudaHostUnregister(hout[i]);
		hipFree(ddata[i]);
		//cudaFree(dout[i]);
		free(hdata[i]);
	}
	hipHostUnregister(refsig);
	hipFree(drefsig);
}
//
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void hIFFT(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        //
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//
	dimGrid.x = grid1;
	//
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //
	printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	//
	vector<hipFloatComplex*> ddata,dout,drout;
	ddata.resize(dsize);
	dout.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&drout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		hipStreamSynchronize(optisdr_streams[i]);
		optisdrifftscale<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],drout[i],ftpoint,chunk);
		//
		hipMemcpyAsync(hout[i],drout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		hipFree(drout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	//
}
//
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
//
void dIFFT(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> dout, int dsize,int chunk, int ftpoint)
{
	//
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        //
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//
	dimGrid.x = grid1;
	//
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //
	printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	//
	vector<hipFloatComplex*> ddata,drout;
	ddata.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//
	}
	//
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&drout[i],chunk*sizeof(hipFloatComplex));
		//
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],drout[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		//cudaStreamSynchronize(optisdr_streams[i]);
		optisdrifftscale<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(drout[i],dout[i],ftpoint,chunk);
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipFree(ddata[i]);
		hipFree(drout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	//
}
//
//
void ddIFFT(vector<hipFloatComplex*> ddata, vector<hipFloatComplex*> dout, int dsize,int chunk, int ftpoint)
{
	//
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        //
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//
	dimGrid.x = grid1;
	//
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //
	printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	//
	vector<hipFloatComplex*> drout;
	//ddata.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],drout[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		//cudaStreamSynchronize(optisdr_streams[i]);
		optisdrifftscale<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(drout[i],dout[i],ftpoint,chunk);
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipFree(ddata[i]);
		hipFree(drout[i]);
		hipfftDestroy(plans[i]);
	}
	//
}
//	
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void dhFFT(vector<hipFloatComplex*> ddata, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	//
	vector<hipFloatComplex*> dout;
	//
	dout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		//cudaHostRegister(hdata[i],chunk*sizeof(cuFloatComplex),cudaHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		//cudaMalloc((void**)&ddata[i],chunk*sizeof(cuFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		//cudaMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(cuFloatComplex),cudaMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hout[i]);
		hipFree(dout[i]);
		hipFree(ddata[i]);
		hipfftDestroy(plans[i]);
	}
	//
}
//
void hFFT(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	//
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	//
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void ddFFT(vector<hipFloatComplex*> ddata, vector<hipFloatComplex*> dout, int dsize,int chunk, int ftpoint)
{
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	//
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipFree(ddata[i]);
		//
		hipfftDestroy(plans[i]);
	}
	//
}
//
void dFFT(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> dout, int dsize,int chunk, int ftpoint)
{
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	//
	vector<hipFloatComplex*> ddata;
	ddata.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//
	}
	//
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		//
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipFree(ddata[i]);
		//cudaFree(dout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	//
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void dhIFFT(vector<hipFloatComplex*> ddata, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        //
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//
	dimGrid.x = grid1;
	//
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //
	printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	//
	vector<hipFloatComplex*> dout,drout;
	//ddata.resize(dsize);
	dout.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		//cudaHostRegister(hdata[i],chunk*sizeof(cuFloatComplex),cudaHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&drout[i],chunk*sizeof(hipFloatComplex));
		//
		//
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Inverse transform fail.\n");}
		//
		//cudaStreamSynchronize(optisdr_streams[i]);
		optisdrifftscale<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],drout[i],ftpoint,chunk);
		//
		hipMemcpyAsync(hout[i],drout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		hipFree(drout[i]);
		
		hipfftDestroy(plans[i]);
	}
	//
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void XCorrSP(vector<hipFloatComplex*> hdata, hipFloatComplex* refsig, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	//printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	hipFloatComplex *drefsig,*dftrefsig;
	hipMalloc((void**)&drefsig,chunk*sizeof(hipFloatComplex));
	hipMalloc((void**)&dftrefsig,chunk*sizeof(hipFloatComplex));
	//cudaHostRegister(refsig,chunk*sizeof(cuFloatComplex),cudaHostRegisterPortable);
	hipMemcpy(drefsig,refsig,chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice);
	//cudaMalloc((void**)&drout,chunk*sizeof(cuFloatComplex));
	//
	vector<hipFloatComplex*> ddata,dout,drout;
	ddata.resize(dsize);
	dout.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//cudaStreamCreate(&optisdr_streams[i]);
		//cudaMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(cuFloatComplex),cudaMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&drout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	
	res = hipfftExecC2C(plans[0],drefsig,dftrefsig,HIPFFT_FORWARD);
	if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//cufftExecC2C(plans[i],ddata[i],dout[i],CUFFT_FORWARD);
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		complexvector_multiply1d1d<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],dftrefsig,drout[i],chunk);
		//
		res = hipfftExecC2C(plans[i],drout[i],ddata[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Inverse transform fail.\n");}
		//
		// TODO: Try using the Block Size =128, i.e optisdrifftscale<<<chunk/128,128,0,optisdr_treams[i]>>>(...);
        	optisdrifftscale<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(ddata[i],dout[i],ftpoint,chunk);
		//
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		hipFree(drout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	hipHostUnregister(refsig);
	hipFree(drefsig);
	//gettimeofday(&t2, 0);
	//double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	//double time2 = ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000000.0;
	//printf("\n[SDR_DSL_INFO]$ Exec. Time for StreamProcessed FFT = %f s ~= %f...!\n", time,time2);
	//cudaDeviceReset();
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void hcomplexvector_multiply(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int index = 0;
	//
	while(index < N)
	{
		outp[index].x = (_cx[index].x*_cy[index].x) - (_cx[index].y*_cy[index].y);//cuCmulf(_cx[index],_cy[index]);
		outp[index].y = (_cx[index].x*_cy[index].y) + (_cx[index].y*_cy[index].x);
		//outp[index] = cuCmulf(_cx[index],_cy[index]);
		index+=1;
	}
}
//
//
void hcomplexvector_multiply2(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N,int ftpoint)
{
	int index = 0;
	//
	printf("\n");
	while(index < ftpoint)
	{
		//outp[index].x = (_cx[index].x*_cy[index].x) - (_cx[index].y*_cy[index].y);//cuCmulf(_cx[index],_cy[index]);
		//outp[index].y = (_cx[index].x*_cy[index].y) + (_cx[index].y*_cy[index].x);
		//printf("[Multiplying: A[%d] = (%f + %f) with B[%d] = (%f + %f) ], ",index,_cx[index].x,_cx[index].y,index,_cy[index].x,_cy[index].y);
		outp[index] = hipCmulf(_cx[index],_cy[index]);
		index+=1;
	}
	printf("\n");
}
//
void complexvector_multiply_h(vector<hipFloatComplex*> inp1,hipFloatComplex* inp2,vector<hipFloatComplex*> outp,int dsize,int N)
{
	int i = 0;
	//
	while(i < dsize)
	{
		hcomplexvector_multiply(inp1[i],inp2,outp[i],N);
		i+=1;
	}
}
//
void complexvector_multiply_h2(vector<hipFloatComplex*> inp1,hipFloatComplex* inp2,vector<hipFloatComplex*> outp,int dsize,int N,int ftpoint)
{
	int i = 0;
	//
	while(i < dsize)
	{
		hcomplexvector_multiply2(inp1[i],inp2,outp[i],ftpoint,ftpoint);
		i+=1;
	}
}
//
//
//
// Creating a Dummy h vector for computing the Hilbert Transform
//
hipFloatComplex* getHilbertHVector(int _dlen)
{
	//
	hipFloatComplex *h = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	//
	h[0] = make_hipFloatComplex(1.0f,0.0f);
	h[_dlen/2] = make_hipFloatComplex(1.0f,0.0f);
	//
	int i = 1;
	int j = (_dlen/2)+1;
	//
	while(i<(_dlen/2))
	{
		h[i] = make_hipFloatComplex(2.0f,0.0f);
		i = i + 1;
		h[j] = make_hipFloatComplex(0.0f,0.0f);
		j = j + 1;
	}
	return h;
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
void HilbertSP(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock;//, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	//blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	//printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//	
	hipFloatComplex* h = resizeVector(getHilbertHVector(ftpoint),ftpoint,chunk);
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	int sigLen = chunk/ftpoint; // Make sure chunk is multiple of 2, better done at DSL level
	//
	hipFloatComplex *dh;
	hipMalloc((void**)&dh,chunk*sizeof(hipFloatComplex));
	hipHostRegister(h,chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
	//cudaMalloc((void**)&drout,chunk*sizeof(cuFloatComplex));
	//
	vector<hipFloatComplex*> ddata,dout,dhout,drout;
	ddata.resize(dsize);
	dout.resize(dsize);
	dhout.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//cufftPlan1d(&plans[i],chunk,CUFFT_C2C,1);
		//cufftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,CUFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,sigLen);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		// TODO: Check Out cudaMallocHost(...), cudaMallocManage(...), ...
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable); // Page-Locked Mem.
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//cudaStreamCreate(&optisdr_streams[i]);
		//cudaMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(cuFloatComplex),cudaMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	hipMemcpyAsync(dh,h,chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[0]);
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dhout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&drout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		complexvector_multiply1d1d<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],dh,dhout[i],chunk);
		//
		res = hipfftExecC2C(plans[i],dhout[i],drout[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		hipMemcpyAsync(hout[i],drout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		hipFree(dhout[i]);
		hipFree(drout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	hipHostUnregister(h);
	hipFree(dh);
	//
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
//
int main()
{
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	//%%%%%%%%% Fill a cudaDeviceProp structure with the properties %%%%%%%%%%%%
	//%%%%%%%%% we need our device to have %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	hipDeviceProp_t deviceProp;
	//cudaEvent_t start, stop;
	struct timeval t1, t2,strt,endt;
	//%%%%%%%%%%%%%% otherwise use device with highest Gflops/s %%%%%%%%%%%%%%%%
	int cuda_device = 0;
	//
	hipGetDevice(&cuda_device); // Get Just the CUDA Device name
	hipSetDevice(cuda_device);
	hipGetDeviceProperties(&deviceProp, cuda_device);
	printf("[SDR_DSL_INFO]$ Device ID :[%d] Name:[%s]\n",cuda_device, deviceProp.name);
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	//%%%%%%%%%%%%%%%% My Main code starts here %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	//%%% TODO: 20*1300*2048 = 53248000 = maximum optimized for..loop size. %%%
	//%%% TODO: 
	int dsize = 5; // Maximum for 1300*2048 sp sizes
	int subchunk = 130000/N_SIGS/dsize;
	int chunk = N_SIGS*SIG_LEN; //1300*2048, lets try 6500*2048
	int ftpoint = 2*SIG_LEN; // TODO: Cater for Zero-Padding
	int dataLen = subchunk*dsize*chunk;
	int rssize = 300;
	//vector<cuFloatComplex*> hdata,hout;
	//hdata.resize(dsize);
	//hout.resize(dsize);
	//
	hipFloatComplex* refsig = resizeVector(getReferenceSignal(rssize,ftpoint),ftpoint,(ftpoint/SIG_LEN)*chunk); // From 300 to chunk,
	//cuFloatComplex* refsig2 = resizeVector(getReferenceSignal(rssize,ftpoint), ftpoint, ftpoint);
	vector< vector<hipFloatComplex*> > hdata2,hout2,hout3; // to cater for 0...2048*130000
	hdata2.resize(subchunk);
	hout2.resize(subchunk);
	hout3.resize(subchunk);
	//
	// Read NetRAD Samples Here
	//
	printf("\n[SDR_DSL_INFO]$ Reading NetRAD Data of Size: %i, subchunk = %i .\n",dataLen,subchunk);
	gettimeofday(&strt, 0);
	//
	vector<short> vsNetRadSamples;
	readNetRadSamples("/media/201x_NetRad/ZA_Trials_2011_06/radarData/2011-06-04/e11_06_04_1740_34_P1_1_130000_S0_1_2047_node3.bin",dataLen,vsNetRadSamples);
	// Format Data Into Matrix
	/*for(int i=0; i<dsize; i++)
	{
		int to=i*chunk+chunk, from=i*chunk;
		hdata[i] = getComplexSamples(vsNetRadSamples,from,to); //0...2047, 2048...4095, etc.
		hout[i] = getComplexEmpty(chunk); // Init. output vector
		//printf("Test::%f",hdata[i][0].x); // Testing
	}*/
	
	for(int j=0; j<subchunk; j++)
	{
		int dfrom = dsize*j;
		int dto = dsize*j+dsize;
		vector<hipFloatComplex*> hdata,hout,houtb;
		hdata.resize(dsize);
		hout.resize(dsize);
		houtb.resize(dsize);
		//
		for(int i=dfrom; i<dto; i++)
		{
			int to=i*chunk+chunk, from=i*chunk;
			//hdata[i%dsize] = getComplexSamples(vsNetRadSamples,from,to); //0...2047, 2048...4095, etc.
			hdata[i%dsize] = getComplexSamples(vsNetRadSamples,from,to,SIG_LEN,ftpoint);
			hout[i%dsize] = getComplexEmpty(chunk,SIG_LEN,ftpoint); // Init. output vector
			houtb[i%dsize] = getComplexEmpty(chunk,SIG_LEN,ftpoint);
			//hout[i%dsize] = getComplexEmpty(chunk); // Init. output vector
			//printf("Test::%f",hdata[i][0].x); // Testing
		}
		hdata2[j] = hdata;
		hout2[j]  = hout;
		hout3[j]  = houtb;
	}
	//
	hipFloatComplex* refsig3 = (hipFloatComplex*)malloc(ftpoint*sizeof(hipFloatComplex));
	for(int j = 0; j< ftpoint; j++)
	{
		refsig3[j] = refsig[j];
	}
	//
	//
	gettimeofday(&endt, 0);	
	double dactime = (1000000.0*(endt.tv_sec-strt.tv_sec) + endt.tv_usec-strt.tv_usec)/1000000.0;
	printf("\n[SDR_DSL_INFO]$ Time Taken by Data Reader = %f s.\n",dactime);	
	//
	//
	gettimeofday(&t1, 0);
	//
	// Exec. Stream Processor
    	//streamprocessor(hdata,hout,dsize,chunk);
	//streamprocessor(hdata,hout,dsize,chunk,ftpoint);
	//for(int i = 0; i<subchunk; i++)
	//{
	//	streamprocessor(hdata2[i],hout2[i],dsize,chunk,ftpoint);
	//}
	for(int i = 0; i<subchunk/2; i++)
	{
		XCorrSP(hdata2[i],refsig,hout2[i],dsize,(ftpoint/SIG_LEN)*chunk,ftpoint);
		//complexvector_multiply_h2(hout2[i],refsig3,hout3[i],dsize,(ftpoint/SIG_LEN)*chunk,ftpoint);
		// HilbertSP(hdata2[i],hout2[i],dsize,(ftpoint/SIG_LEN)*chunk,ftpoint);
	}
	//complexvector_multiply_h2(hout2[0],refsig3,hout3[0],dsize,(ftpoint/SIG_LEN)*chunk,ftpoint);
		
	hipSetDevice(cuda_device+1);
	hipGetDeviceProperties(&deviceProp,cuda_device+1);
	printf("[SDR_DSL_INFO]$ Setting Device ID :[%d] Name:[%s]\n",cuda_device+1, deviceProp.name);
	//
	for(int i = subchunk/2; i<subchunk; i++)
	{
		//dFFT(hdata2[i],dout2[i],dsize,(ftpoint/SIG_LEN)*chunk,ftpoint);
		XCorrSP(hdata2[i],refsig,hout2[i],dsize,(ftpoint/SIG_LEN)*chunk,ftpoint);
		//complexvector_multiply_h2(hout2[i],refsig3,hout3[i],dsize,(ftpoint/SIG_LEN)*chunk,ftpoint);
		//HilbertSP(hdata2[i],hout2[i],dsize,(ftpoint/SIG_LEN)*chunk,ftpoint);
	}
	//
	gettimeofday(&t2, 0);
	double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	double time2 = ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000000.0;
	printf("\n[SDR_DSL_INFO]$ Time Taken by FFT = %f s ~= %f...!\n", time,time2);
	//
     	float *fftOutput = (float*)malloc(chunk*sizeof(float));
      	for (int i = 0; i < 1; i++)//subchunk
      	{
		hipFloatComplex* tmpout = hout2[0][0];//hout2[0][0];//hout2[0][i];//TODO: Uncomment this after testing 
		//cuFloatComplex *tmpout = hdata2[0][i];
		for(int j = 0; j< ftpoint; j++)
		{
			fftOutput[j] = hipCrealf(tmpout[j]);
		}
		printf("[SDR_DSL_INFO]$ Output = [");
		for( int i = 0; i < ftpoint;i++)
		{
			printf("%f , ",fftOutput[i]);
		}
		printf("]\n");
      	}
	//
	// Clean-Up Boy
	for(int j=0;j<subchunk;j++)
	{
		for( int i = 0; i < dsize;i++)
		{
			free(hout2[j][i]); // Clear Host Memory
			//cudaFree(dout2[j][i]);
		}
	}
	//cudaThreadExit();
	//
	free(refsig);
	free(fftOutput);
	hipDeviceReset();
	//
	return 0;
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%% Example Test Signals %%%%%%%%%%%%%%%%%%%%%%%%%%%%%
void getShortSamples(vector<short> &vSamples, unsigned int nSamples)
{
	//Read
	vSamples.resize(nSamples);
	for(unsigned int i = 0; i< nSamples; i++)
	{
		vSamples[i] = i;
		//printf("%i,",i);
	}
}
//
void getRealSamples(vector<float> &vSamples, unsigned int nSamples)
{
	//Read
	vSamples.resize(nSamples);
	for(unsigned int i = 0; i< nSamples; i++)
	{
		vSamples[i] = (float)(i);
		//printf("%i,",i);
	}
}
//
float* getRealFSamples(int dalen)
{
	//Read
	float *dSamples = (float *)malloc(dalen*sizeof(float));
	for(int i = 0; i< dalen; i++)
	{
		dSamples[i] = (float)(i);
		//printf("%f,",dSamples[i]);
	}
	return dSamples;
}
//
void getRandSamples(vector<float> &vSamples, unsigned int nSamples, int nbound)
{
	//Read
	vSamples.resize(nSamples);
	for(unsigned int i = 0; i< nSamples; i++)
		vSamples[i] = rand()%nbound;
}
//
hipFloatComplex* getComplexSamples(int _dlen)
{
	hipFloatComplex *dSamples = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	for(int i = 0; i< _dlen; i++)
	{
		dSamples[i] = make_hipFloatComplex((float)(i),(float)(i));
	}
	return dSamples;
}
hipFloatComplex* getComplexRandomSamples(int _dlen)
{
	hipFloatComplex *dSamples = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	for(int i = 0; i< _dlen; i++)
	{
		dSamples[i] = make_hipFloatComplex(rand(),rand());
	}
	return dSamples;
}
//
void getAscendingRandSamples(vector<float> &vSamples, unsigned int nSamples, int nbound)
{
	//Read
	vSamples.resize(nSamples);
	srand(time(0));
	for(unsigned int i = 0; i< nSamples; i++)
		vSamples[i] = rand()%nbound;
}
//%%%%%%%%%%%%%%%%%%%%%%% End of FIR Filter Implementation %%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
