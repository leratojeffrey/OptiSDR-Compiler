#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <assert.h>

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}

/*********************/
/* CUFFT ERROR CHECK */
/*********************/
static const char *_cudaGetErrorEnum(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";

        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";

        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";

        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";

        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";

        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";

        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";

        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}

#define cufftSafeCall(err)      __cufftSafeCall(err, __FILE__, __LINE__)
inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
{
    if( HIPFFT_SUCCESS != err) {
                fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
                           _cudaGetErrorEnum(err)); \
             hipDeviceReset(); assert(0); \
    }
}

/********/
/* MAIN */
/********/
void main() {

    hipfftHandle forward_plan, inverse_plan; 

    int batch = 3;
    int rank = 2;

    int nRows = 5;
    int nCols = 5;
    int n[2] = {nRows, nCols};

    int idist = nRows*nCols;
    int odist = nRows*(nCols/2+1);

    int inembed[] = {nRows, nCols};
    int onembed[] = {nRows, nCols/2+1};

    int istride = 1;
    int ostride = 1;

    cufftSafeCall(hipfftPlanMany(&forward_plan,  rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch));

    float *h_in = (float*)malloc(sizeof(float)*nRows*nCols*batch);
    for(int i=0; i<nRows*nCols*batch; i++) h_in[i] = 1.f;

    float2* h_freq = (float2*)malloc(sizeof(float2)*nRows*(nCols/2+1)*batch);

    float* d_in;            gpuErrchk(hipMalloc(&d_in, sizeof(float)*nRows*nCols*batch)); 
    float2* d_freq; gpuErrchk(hipMalloc(&d_freq, sizeof(float2)*nRows*(nCols/2+1)*batch)); 

      gpuErrchk(hipMemcpy(d_in,h_in,sizeof(float)*nRows*nCols*batch,hipMemcpyHostToDevice));

    cufftSafeCall(hipfftExecR2C(forward_plan, d_in, d_freq));

    gpuErrchk(hipMemcpy(h_freq,d_freq,sizeof(float2)*nRows*(nCols/2+1)*batch,hipMemcpyDeviceToHost));

    for(int i=0; i<nRows*(nCols/2+1)*batch; i++) printf("Direct transform: %i %f %f\n",i,h_freq[i].x,h_freq[i].y); 

    cufftSafeCall(hipfftPlanMany(&inverse_plan, rank, n, onembed, ostride, odist, inembed, istride, idist, HIPFFT_C2R, batch));

    cufftSafeCall(hipfftExecC2R(inverse_plan, d_freq, d_in));

    gpuErrchk(hipMemcpy(h_in,d_in,sizeof(float)*nRows*nCols*batch,hipMemcpyDeviceToHost));

    for(int i=0; i<nRows*nCols*batch; i++) printf("Inverse transform: %i %f \n",i,h_in[i]); 

    getchar();

}
