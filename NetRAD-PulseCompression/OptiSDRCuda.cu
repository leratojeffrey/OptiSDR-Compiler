#include "hip/hip_runtime.h"
#ifndef _OPTISDR_CUDA_
#define _OPTISDR_CUDA_
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
#include "OptiSDRCuda.h"
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%% OptiSDR Suppot Functions %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%
//%%%%%% Read Text File to Binary file %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
int readToBinary(string fln1, string fln2)
{
	std::ifstream in(fln1.c_str()); // Create a simple Text File Reader - C++
	std::ofstream out(fln2.c_str(),std::ios::binary); // Create a simple C++ Binary File Writer

	int d; // We read as Integers, well we can make short if we want
	while(in >> d) // Untill end of file
	{
		out.write((char*)&d, sizeof d); // Write into output binary file
		//cout<<d<<endl; // Uncomment to display values
	}
	// Good, now we have a .bin file that GNURadio can read.
	return 0;
}
//%
void readNetRADTextFile(string fln,vector<short> &vsSamples, int N)
{
	vsSamples.resize(N);
	std::ifstream in(fln.c_str()); // Create a simple Text File Reader - C++
	//
	int i = 0;
	short d; // We read as Short
	while(in >> d) // Visit every element untill end of file
	{
		vsSamples[i] = d;
		i++;
		//out.write((char*)&d, sizeof d); // Write into output binary file
		//cout<<d<<endl; // Uncomment to display values
	}
	cout<<endl<<i<<endl;
	// Good, done..
}
//%
//%%%
//%
//double GPUcalTime = 0.0;
void readNetRadSamples(string strFilename, unsigned int uiNSamples, vector<short> &vsSamples)
{
	//Read
	ifstream oIFS;
	oIFS.open(strFilename.c_str(), ifstream::binary);
	if(!oIFS.is_open())
	{	
		cout << "[SDR_DSL_INFO]$ Error unable to open file \"" << strFilename << "\"" << endl;
		oIFS.close();
		exit(1);
	}
	//
	vsSamples.resize(uiNSamples);
	//
	oIFS.read((char*)&vsSamples.front(), sizeof(short) * uiNSamples);

	if(oIFS.gcount() << sizeof(short) * uiNSamples && oIFS.eof())
	{
		cout << "[SDR_DSL_INFO]$ Warning: hit end of file after " << oIFS.gcount() / sizeof(short) << " samples. Output is shortened accordingly." << endl;
		vsSamples.resize(oIFS.gcount() / sizeof(short));
	}
	//
	oIFS.close();
}
//
void readFileToVector(string strFilename, unsigned int numSamples, vector<short> &vsSamples)
{
	//Read File into Complex Data
	//
	vsSamples.resize(numSamples);	
	int dataSize=numSamples*sizeof(short);
	short *buffer = (short*)malloc(dataSize);
	//
	struct timeval t3,t4;
	gettimeofday(&t3, 0);
	FILE *fileIO;
	fileIO=fopen(strFilename.c_str(),"r");
	if(!fileIO)
	{
		printf("[SDR_DSL_ERROR]$ Unable to open file!");
		exit(1);
	}	
	fread(buffer,dataSize,1,fileIO); // Serial File I/O: Read data into buffer
	fclose(fileIO);
	//
	gettimeofday(&t4, 0);
	double time1 = (1000000.0*(t4.tv_sec-t3.tv_sec) + t4.tv_usec-t3.tv_usec)/1000000.0;
	printf("[SDR_DSL_INFO]$ Overall Reading Data Into Memory = %f s .\n", time1);
	//
	#pragma omp parallel for
	for(int i=0; i<numSamples; i++)
	{
		vsSamples[i] = buffer[i];
	}
	//
	free(buffer);
	//
}
//
void readFileToComplex(string strFilename, unsigned int numSamples, hipFloatComplex *outVec, int type)
{
	//Read File into Complex Data
	outVec = (hipFloatComplex*)malloc(numSamples*sizeof(hipFloatComplex));
	if(type == 1)
	{
		int dataSize=numSamples*sizeof(short);
		short *buffer = (short*)malloc(dataSize);
		FILE *fileIO;
		fileIO=fopen(strFilename.c_str(),"r");
		if(!fileIO)
		{
			printf("[SDR_DSL_ERROR]$ Unable to open file!\n");
			exit(1);
		}	
		fread(buffer,dataSize,1,fileIO); // Serial File I/O: Read data into buffer
		fclose(fileIO);
		//
		#pragma omp parallel for
		for(int i=0; i<numSamples; i++)
		{
			outVec[i] = make_hipFloatComplex((float)buffer[i],0.0f);
		}
		//
		free(buffer);
	}
	else if(type == 2)
	{
		int dataSize=numSamples*sizeof(float);
		float *buffer = (float*)malloc(dataSize);
		FILE *fileIO;
		fileIO=fopen(strFilename.c_str(),"r");
		if(!fileIO)
		{
			printf("[SDR_DSL_ERROR]$ Unable to open file!");
			exit(1);
		}	
		fread(buffer,dataSize,1,fileIO); // Serial File I/O: Read data into buffer
		fclose(fileIO);
		//
		#pragma omp parallel for
		for(int i=0; i<numSamples; i++)
		{
			outVec[i] = make_hipFloatComplex(buffer[i],0.0f);
		}	
		//
		free(buffer);
	}
	else if(type == 3)
	{
		int dataSize=numSamples*sizeof(double);
		double *buffer = (double*)malloc(dataSize);
		FILE *fileIO;
		fileIO=fopen(strFilename.c_str(),"r");
		if(!fileIO)
		{
			printf("[SDR_DSL_ERROR]$ Unable to open file!");
			exit(1);
		}	
		fread(buffer,dataSize,1,fileIO); // Serial File I/O: Read data into buffer
		fclose(fileIO);
		//
		#pragma omp parallel for
		for(int i=0; i<numSamples; i++)
		{
			outVec[i] = make_hipFloatComplex((float)buffer[i],0.0f);
		}	
		//
		free(buffer);
	}
	else
	{
		int dataSize=numSamples*sizeof(int);
		int *buffer = (int*)malloc(dataSize);
		FILE *fileIO;
		fileIO=fopen(strFilename.c_str(),"r");
		if(!fileIO)
		{
			printf("[SDR_DSL_ERROR]$ Unable to open file!");
			exit(1);
		}	
		fread(buffer,dataSize,1,fileIO); // Serial File I/O: Read data into buffer
		fclose(fileIO);
		//
		#pragma omp parallel for
		for(int i=0; i<numSamples; i++)
		{
			outVec[i] = make_hipFloatComplex((float)buffer[i],0.0f);
		}	
		//
		free(buffer);
	}
}
//
//
void readFileToFloat(string strFilename, unsigned int numSamples, float *outVec)
{
	//Read File into Complex Data
	outVec = (float*)malloc(numSamples*sizeof(float));
	int dataSize=numSamples*sizeof(float);
	//
	FILE *fileIO;
	fileIO=fopen(strFilename.c_str(),"r");
	if(!fileIO)
	{
		printf("[SDR_DSL_ERROR]$ Unable to open file!");
		exit(1);
	}	
	fread(outVec,dataSize,1,fileIO); // Serial File I/O: Read data into buffer
	fclose(fileIO);
	//
}
//
//
hipFloatComplex* getComplexSamples(vector<short> &vSamples, int _dlen)
{
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	for(int i = 0; i< _dlen; i++)
	{
		cSamples[i] = make_hipFloatComplex(vSamples[i],0.0);
	}
	return cSamples;
}
//
hipFloatComplex* getComplexSamples(vector<short> &vSamples, int from, int to)
{
	int dlen = to-from;
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(dlen*sizeof(hipFloatComplex));
	for(int i = from; i< to; i++)
	{
		cSamples[i%dlen] = make_hipFloatComplex(vSamples[i],0.0);
	}
	return cSamples;
}
//
//
hipFloatComplex* getComplexEmpty(int _dlen)
{
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	for(int i = 0; i< _dlen; i++)
	{
		cSamples[i] = make_hipFloatComplex(0.0,0.0);
	}
	return cSamples;
}
//
void readNetRadSamples2(string strFilename, unsigned int nsamples, vector<float> &vsSamples)
{
	//Read
	ifstream oIFS;
	oIFS.open(strFilename.c_str(),ifstream::in);
	if(!oIFS.is_open())
	{	
		cout << "[SDR_DSL_INFO]$ Error unable to open file \"" << strFilename << "\"" << endl;
		oIFS.close();
		exit(1);
	}
	//
	vsSamples.resize(nsamples);
	int i = 0;
	while(i < nsamples)
	{
		oIFS>>vsSamples[i];
		//vsSamples[i] = atof(tmp[i].c_str());
		//printf("%f, ",vsSamples[i]);
		i++;
	}
	oIFS.close();
}
//
hipFloatComplex* getReferenceSignal(int _dlen)
{
	hipFloatComplex *refsig = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	//
	vector<float> rSamples,iSamples;
	readNetRadSamples2("data/rref3sig2.dat",_dlen,rSamples);
	readNetRadSamples2("data/iref3sig2.dat",_dlen,iSamples);
	//
	for(int i = 0; i< _dlen; i++)
	{
		refsig[i] = make_hipFloatComplex(rSamples[i],iSamples[i]);
	}
	return refsig;
}
//
//
hipFloatComplex* getReferenceSignal(string fnamer, string fnamei, int _dlen)
{
	hipFloatComplex *refsig = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	//
	vector<float> rSamples,iSamples;
	readNetRadSamples2(fnamer,_dlen,rSamples);
	readNetRadSamples2(fnamei,_dlen,iSamples);
	//
	for(int i = 0; i< _dlen; i++)
	{
		refsig[i] = make_hipFloatComplex(rSamples[i],iSamples[i]);
	}
	//printf("\n\n%f\n\n\n",refsig[_dlen-1]);
	return refsig;
}
//
//
//
hipFloatComplex* getReferenceSignal(string fnamer, string fnamei, int _dlen, int ftpoint)
{
	hipFloatComplex *refsig = (hipFloatComplex *)malloc(ftpoint*sizeof(hipFloatComplex));
	//
	vector<float> rSamples,iSamples;
	readNetRadSamples2(fnamer,_dlen,rSamples);
	readNetRadSamples2(fnamei,_dlen,iSamples);
	//
	for(int i = 0; i< _dlen; i++)
	{
		refsig[i] = make_hipFloatComplex(rSamples[i],iSamples[i]);
	}
	for(int i = _dlen; i< ftpoint; i++)
	{
		refsig[i] = make_hipFloatComplex(0.0f,0.0f);
	}
	//printf("\n\n%f\n\n\n",refsig[_dlen-1]);
	return refsig;
}
//
//
// This one accepts the return size and file size.
//
hipFloatComplex* getReferenceSignal(int fsize, int _dlen)
{
	hipFloatComplex *refsig = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	//
	//
	vector<float> rSamples,iSamples;
	readNetRadSamples2("data/rref3sig2.dat",fsize,rSamples);
	readNetRadSamples2("data/iref3sig2.dat",fsize,iSamples);
	//
	for(int i = 0; i< fsize; i++)
	{
		refsig[i] = make_hipFloatComplex(rSamples[i],iSamples[i]);
		// 
	}
	//
	for(int j = (_dlen-fsize); j< _dlen; j++)
	{
		refsig[j] = make_hipFloatComplex(0.0f,0.0f);
	}
	//
	return refsig;
}
//
hipFloatComplex* resizeVector(hipFloatComplex *inp, int oldlen, int newlen)
{
	hipFloatComplex *outp = (hipFloatComplex*)malloc(newlen*sizeof(hipFloatComplex));
	//
	for(int i = 0; i< newlen; i++)
	{
		outp[i] = inp[i%oldlen];
	}
	//
	//outp[0] = make_hipFloatComplex(2.0f,2.0f); // Just for testing: TODO: Remove when done
	//outp[2048] = make_hipFloatComplex(2.0f,2.0f); // Just for testing: TODO: Remove when done
	//outp[4096] = make_hipFloatComplex(2.0f,2.0f); // Just for testing: TODO: Remove when done
	return outp;
}

//
hipFloatComplex* getComplexSamples(vector<short> &vSamples, int from, int to, int chunksize, int outputsize)
{
	int dlen = outputsize*((to-from)/chunksize);
	int skip = outputsize-chunksize;
	//
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(dlen*sizeof(hipFloatComplex));
	//for(int i = 0;i<dlen;i++) cSamples[i] = make_hipFloatComplex(0.0f,0.0f);
	int idx = 0;
	if(skip>0)
	{
		for(int i = from; i< to; i++)
		{
		
			if(((i%chunksize)==0)&&(i>from))
			{
				idx+=skip; // We skip 
				cSamples[idx] = make_hipFloatComplex((float)vSamples[i],0.0f);
				idx+=1;
			}
			else
			{
				cSamples[idx] = make_hipFloatComplex((float)vSamples[i],0.0f);
				idx+=1;
			}
		}
	}
	return cSamples;
}
//
hipFloatComplex* getComplexEmpty(int _dlen, int chunksize, int outputsize)
{
	int dlen = (_dlen)*(outputsize/chunksize);
	//
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(dlen*sizeof(hipFloatComplex));
	for(int i = 0;i<dlen;i++) cSamples[i] = make_hipFloatComplex(0.0f,0.0f);
	//
	return cSamples;
}
//
hipFloatComplex* getChunk(vector<short> &inp, int from, int to) // Want to get complex values from shorts vector
{
	//
	int chunk = to - from;
	hipFloatComplex *outp = (hipFloatComplex*)malloc(chunk*sizeof(hipFloatComplex));
	//	
	int index = 0;
	for(int i = from; i< to; i++)
	{
		outp[index] = make_hipFloatComplex((float)inp[i],0.0f);
		index+=1;
	}
	//
	//
	return outp;
}
//
//
hipFloatComplex* getChunk(hipFloatComplex *inp, int from, int to)
{
	//
	int chunk = to-from;
	hipFloatComplex *outp = (hipFloatComplex*)malloc(chunk*sizeof(hipFloatComplex));
	//
	copy(inp + from,inp + to, outp + 0);
	//
	return outp;
}

//
hipFloatComplex* getChunk(hipFloatComplex *inp, int chunk, int from, int to)
{
	//
	//
	hipFloatComplex *outp = (hipFloatComplex*)malloc(chunk*sizeof(hipFloatComplex));
	//
	//copy(inp + from,inp + to, outp + 0);
	int index = 0;
	for(int i = from; i< to; i++)
	{
		outp[index] = inp[i];
		index+=1;
	}
	//
	return outp;
}
//
hipFloatComplex* getZeroPadded(hipFloatComplex *inp, int initlen, int newlen, int skip)
{
	//
	//
	hipFloatComplex *cSamples = (hipFloatComplex *)malloc(newlen*sizeof(hipFloatComplex));
	//
	int idx = 0;
	if(skip>0)
	{
		for(int i = 0; i< initlen; i++)
		{
		
			if(((i%skip)==0)&&(i>0))
			{
				idx+=skip; // We skip 
				cSamples[idx] = inp[i];
				idx+=1;
			}
			else
			{
				cSamples[idx] = inp[i];
				idx+=1;
			}
		}
	}
	return cSamples;
}
//
void append(hipFloatComplex *A, hipFloatComplex *B, int sizeA, int sizeB, int from)
{
	int index = 0;
	if((sizeA-sizeB)>=sizeB)
	{
		for(int i=from;i<sizeB;i++)
		{
			A[i] = B[index];
			index++;
		}		
		//printf("index = %d\n",index);
	}
}
//
void writeFileF(const char *fpath, float *data,	const unsigned int len)
{
    printf("[SDR_DSL_INFO]$ Output file: %s\n", fpath);
    FILE *fo;

    unsigned int i=0;

    if ( (fo = fopen(fpath, "w")) == NULL) {printf("[SDR_DSL_INFO]$ IO Error\n"); /*return(CUTFalse);*/}

    for (i=0; i<len; ++i)
    {
	if ( (fprintf(fo,"%.7e\n", data[i])) <= 0 )
	{
	    printf("[SDR_DSL_INFO]$ File write Error.\n");
	    fclose(fo);
	    //return(CUTFalse);
	}
    }

    fclose(fo);
    //return(CUTTrue);
}
//
void writeFileF(const char *fpath,hipFloatComplex *xdata, const unsigned int len)
{
    printf("[SDR_DSL_INFO]$ Output file: %s\n", fpath);
    FILE *fo;

    unsigned int i=0;

    if ( (fo = fopen(fpath, "w")) == NULL) {printf("[SDR_DSL_INFO]$ IO Error\n");}

    for (i=0; i<len; ++i)
    {
	//if((fprintf(fo,"%.7e + %.7ei\n",hipCrealf(xdata[i]),hipCimagf(xdata[i]))) <= 0 )
	if((fprintf(fo,"%f \n",hipCabsf(xdata[i]))) <= 0 )
	{
		printf("[SDR_DSL_INFO]$ File write Error.\n");
		fclose(fo);
	}
    }

    fclose(fo);
    //return(CUTTrue);
}
//
void vresize(hipFloatComplex *inp, hipFloatComplex *outp, int inlen, int skip)
{
	int index = 2*skip;
	int iters = inlen/skip;
	//printf("%d",iters);
	for(int i=0; i<iters; i++)
	{
		for(int j=0; j<skip; j++)
		{
			outp[i*index + j] = inp[j];
		}
	}
	//
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%% Complex Multiplication Kernel %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__global__ void complexvector_multiply(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < N)
		outp[tid] = hipCmulf(_cx[tid],_cy[tid]);
}
//
// This Kernel Utilize a 1-D Grid to a 1-D data indexes (1-D BlockDim = 1024 threads)
// We can multiply 16 Million floating point values in Parallel
__global__ void complexvector_multiply1d1d(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	//
	if(index < N)
	{
		outp[index].x = (_cx[index].x*_cy[index].x) - (_cx[index].y*_cy[index].y);//hipCmulf(_cx[index],_cy[index]);
		outp[index].y = (_cx[index].x*_cy[index].y) + (_cx[index].y*_cy[index].x); //
	}
}
// This Kernel Utilize a 2-D Grid flattened to a 1-D data indexes (1-D BlockDim = 1024 threads)
// We can multiply 68.747788288 Million floating point values in Parallel
__global__ void complexvector_multiply2d1d(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
	//
	int xsize = blockDim.x*gridDim.x; // X dimension total number of thread
	//
	int index = tidy*xsize + tidx; // Index through 2-D grid
	//
	if(index < N)
		outp[index] = hipCmulf(_cx[index],_cy[index]);
}
// This Kernel Utilize a 3-D Grid flattened to a 1-D data indexes
// We can multiply 129 Million floating point values in Parallel
__global__ void complexvector_multiply3d1d(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
	int tidz = threadIdx.z + blockIdx.z * blockDim.z;
	//
	int xsize = blockDim.x*gridDim.x; // X dimension total number of thread
	int zsize = xsize*blockDim.y*gridDim.y; // Entire 2-D grid numer of thread
	//
	int xyindex = tidy*xsize + tidx; // Index through 2-D grid
	int index = tidz*zsize + xyindex; // Index through entire 3-D grid
	//
	if(index < N)
		outp[index] = hipCmulf(_cx[index],_cy[index]);
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%% Complex Conjugate Kernel %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__global__ void complexvector_conjugate(hipFloatComplex *_cx, hipFloatComplex *outp,int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		//outp[tid].x= (_cx[tid].x*_cy[tid].x)-(_cx[tid].y*_cy[tid].y);
		//outp[tid].y= (_cx[tid].x*_cy[tid].y)+(_cy[tid].x*_cx[tid].y);
		outp[tid] = hipConjf(_cx[tid]);
		tid += blockDim.x * gridDim.x;
	}
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%% Complex Absolute Kernel %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__global__ void complexvector_abs(hipFloatComplex *_cx,float *outp,int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		//outp[tid].x= (_cx[tid].x*_cy[tid].x)-(_cx[tid].y*_cy[tid].y);
		//outp[tid].y= (_cx[tid].x*_cy[tid].y)+(_cy[tid].x*_cx[tid].y);
		outp[tid] = hipCabsf(_cx[tid]);
		tid += blockDim.x * gridDim.x;
	}
}
//
__global__ void _10logabs(hipFloatComplex *_cx, hipFloatComplex *outp, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid<N)
		outp[tid].x = 10.0f*logf(hipCabsf(_cx[tid]));
}
//
// float Vpp,float div, float logVal
//
__global__ void _20log10fabs(hipFloatComplex *_cx, hipFloatComplex *outp, int N,hipFloatComplex Vpp,hipFloatComplex divV,float logV)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid<N)
	{
		outp[tid].x = logV*log10f(hipCabsf(hipCdivf(hipCmulf(_cx[tid],Vpp),divV))); // logV*log10f(hipCabsf(_cx[tid]));
		outp[tid].y = 0.0f;
	}
}
//
//
__global__ void optisdrifftscale(hipFloatComplex *invec, hipFloatComplex *out, int fp, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < N)
		out[tid] = make_hipFloatComplex(hipCrealf(invec[tid])/(float)fp,hipCimagf(invec[tid])/(float)fp);
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%% Simplified DSP CUDA Kernel Calls %%%%%%%%%%%%%%%%%%%%%%
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%% Interference Filter Using CUDA Stream Processing %%%%%%%%%%%%%
//
// The Interference Filter Kernel
__global__ void ones(hipFloatComplex *outd,int N, int offset)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < N)
		outd[tid+offset] = make_hipFloatComplex(1.0f,1.0f);
}
//
__global__ void zeros(hipFloatComplex *outd,int N, int offset)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < N)
		outd[tid+offset] = make_hipFloatComplex(0.0f,0.0f);
}
//
// Interference Filter Stream Processor
//
//
hipFloatComplex* interFilterCoef(int M, int fIdx)
{
	hipFloatComplex *cSamples = (hipFloatComplex*)malloc(M*sizeof(hipFloatComplex));
	for(int i = 0; i< M; i++)
	{
		cSamples[i] = make_hipFloatComplex(1.0f,1.0f);
	}
	//1:fIdx+1
	for(int i=0;i<fIdx+1;i++) cSamples[i] = make_hipFloatComplex(0.0f,0.0f);
	for(int i=M-fIdx+1;i<M;i++) cSamples[i] = make_hipFloatComplex(0.0f,0.0f);
	return cSamples;
}
//
void _meanSqr(hipFloatComplex* hdata, int M)
{
	float mean= 0.0;
	
	for(int i=0;i<M;i++)
	{
		mean+=pow(hipCabsf(hdata[i]),2);
	}
	float meansqr = sqrt(mean/M);
	for(int i=0;i<M;i++)
	{
		hdata[i].x = hdata[i].x/meansqr;
		hdata[i].y = hdata[i].y/meansqr;
	}
	//return hdata;
}
//
//
double _interferenceFilter(hipFloatComplex* hdata, float fc, float fs, int M) // 
{
	//
	struct timeval kernt1, kernt2;
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock;//, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(M/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	//blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	//printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	float Df = fs/(float)M; //the freuqency step
	int fIdx = (int)ceil(fc/Df); //the index of the cut-off freq, it is set to the closest larger integer
	printf(" %d. \n",fIdx);
	//	
	hipFloatComplex* h = interFilterCoef(M,fIdx);//
	hipfftHandle plan;
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {M};
	int sigLen = M/M; // Make sure chunk is multiple of 2, better done at DSL level
	//
	hipFloatComplex *dh;
	hipMalloc((void**)&dh,M*sizeof(hipFloatComplex));
	//hipHostRegister(h,M*sizeof(hipFloatComplex),hipHostRegisterPortable);
	//hipMalloc((void**)&drout,chunk*sizeof(hipFloatComplex));
	//
	hipFloatComplex* ddata,*dout,*dhout,*drout;
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	res = hipfftPlanMany(&plan, 1, n,
    		NULL, 1,M, //advanced data layout, NULL shuts it off
    		NULL, 1,M, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,sigLen);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
	// Trying to Create Page-Locked std::vector
	// 
	//
	hipMemcpy(dh,h,M*sizeof(hipFloatComplex),hipMemcpyHostToDevice);
	//
	hipMalloc((void**)&ddata,M*sizeof(hipFloatComplex));
	hipMalloc((void**)&dout,M*sizeof(hipFloatComplex));
	hipMalloc((void**)&dhout,M*sizeof(hipFloatComplex));
	hipMalloc((void**)&drout,M*sizeof(hipFloatComplex));
	//
	hipMemcpy(ddata,hdata,M*sizeof(hipFloatComplex),hipMemcpyHostToDevice);
	//
	//
	gettimeofday(&kernt1, 0);
	// Execution
	//
	res = hipfftExecC2C(plan,ddata,dout,HIPFFT_FORWARD);
	if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
	//
	complexvector_multiply1d1d<<<dimGrid,dimBlock>>>(dout,dh,dhout,M);
	//
	res = hipfftExecC2C(plan,dhout,drout,HIPFFT_BACKWARD);
	if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
	//
	optisdrifftscale<<<dimGrid,dimBlock>>>(drout,dout,M,M);
	//
	//
	gettimeofday(&kernt2, 0);
	double GPUTime = (1000000.0*(kernt2.tv_sec-kernt1.tv_sec) + kernt2.tv_usec-kernt1.tv_usec); ///1000000.0;
	//
	hipMemcpy(hdata,dout,M*sizeof(hipFloatComplex),hipMemcpyDeviceToHost);
	// Releasing Computing Resources
	hipFree(ddata);
	hipFree(dout);
	hipFree(dhout);
	hipFree(drout);
	//free(hdata);
	hipfftDestroy(plan);
	hipFree(dh);
	free(h);
	//
	return GPUTime;
}
//

//
double _fft(hipFloatComplex* hdata, int M) // 
{
	//
	struct timeval kernt1, kernt2;
	//
	hipfftHandle plan;
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {M};
	int sigLen = M/M; // Make sure chunk is multiple of 2, better done at DSL level
	//
	//
	//
	//
	hipFloatComplex* ddata,*dout;
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	res = hipfftPlanMany(&plan, 1, n,
    		NULL, 1,M, //advanced data layout, NULL shuts it off
    		NULL, 1,M, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,sigLen);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
	//
	// 
	//
	hipMalloc((void**)&ddata,M*sizeof(hipFloatComplex));
	hipMalloc((void**)&dout,M*sizeof(hipFloatComplex));
	//
	hipMemcpy(ddata,hdata,M*sizeof(hipFloatComplex),hipMemcpyHostToDevice);
	//
	//
	gettimeofday(&kernt1, 0);
	// Execution
	//
	res = hipfftExecC2C(plan,ddata,dout,HIPFFT_FORWARD);
	if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
	//
	//
	gettimeofday(&kernt2, 0);
	double GPUTime = (1000000.0*(kernt2.tv_sec-kernt1.tv_sec) + kernt2.tv_usec-kernt1.tv_usec); ///1000000.0;
	//
	hipMemcpy(hdata,dout,M*sizeof(hipFloatComplex),hipMemcpyDeviceToHost);
	// Releasing Computing Resources
	hipFree(ddata);
	hipFree(dout);
	//free(hdata);
	hipfftDestroy(plan);
	//
	return GPUTime;
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%% Hamming Window using CUDA %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// TODO: Use 2D and 3D to cater for bigger sizes
//
__global__ void _hamming(hipFloatComplex *outp, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid<N)
		outp[tid].x = 0.54f - 0.46f * cos(2.0f*mPI*(tid)/(float)(N-1));
}
//
__global__ void repmat(hipFloatComplex *dhout,hipFloatComplex *drout,int hmsize,int chunks)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid<hmsize)
	{
		//for(int i = 0; i<hmsize; i++)
		//{
			drout[tid] = dhout[tid];
		//}
	}
}
//
void repMat(hipFloatComplex *dhout,hipFloatComplex *drout,int hmsize,int chunks,int padding) // e.g. repMat(in,out,2048,2048*1300,2);
{
	int tid = 0;
	int col = padding*hmsize;
	int row = chunks/hmsize;
	while(tid<row)
	{
		for(int i = 0; i<hmsize; i++)
		{
			drout[tid*col+i] = dhout[i];
		}
		tid++;
	}
}
//
void zepadMat(hipFloatComplex *dhout,hipFloatComplex *drout,int row,int col,int padding) // e.g. zepadMat(in,out,1300,2048,2);
{
	int tid = 0;
	int ncol = padding*col;
	//int row = padding*chunks/hmsize;
	while(tid<row)
	{
		for(int i = 0; i<col; i++)
		{
			drout[tid*ncol+i] = dhout[tid*col+i];
		}
		tid++;
	}
}
//
hipFloatComplex* hamming(int N)
{
	hipFloatComplex *outp = getComplexEmpty(N);
	int tid = 0;
	while(tid<N)
	{
		outp[tid].x = 0.54f - 0.46f * cos(2.0f*mPI*(tid)/(float)(N-1));
		tid++;
	}
	return outp;
}
//
//
hipFloatComplex* hamming(hipFloatComplex *inp,int N,int ftpoint)//
{
	hipFloatComplex *outp = getComplexEmpty(ftpoint);
	int tid = 0;
	while(tid<N)
	{
		outp[tid].x = 0.54f - 0.46f * cos(2.0f*mPI*(tid)/(float)(N-1));
		outp[tid] = hipCmulf(inp[tid],outp[tid]);
		tid++;
	}
	for(int i=N;i<ftpoint;i++) outp[i]=make_hipFloatComplex(0.0f,0.0f);
	return outp;
}
//
//
void TestStuff()
{
	//int cuda_device=0;
	//
	////hipGetDevice(&cuda_device); // Get Just the CUDA Device name
	//hipSetDevice(cuda_device);
	struct timeval t1,t2;
	//
	int ftpoint = 2048;
	int chunk = ftpoint*1300;
	//dim3 dimBlock, dimGrid;
    	//int threadsPerBlock, blocksPerGrid;
	//dimBlock.x = 1024;
	//threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
	//int grid1 = ceil(chunk/(float)threadsPerBlock);
	//dimGrid.x = grid1;
	//blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //
	//
	gettimeofday(&t1, 0);
	hipFloatComplex *khout = getComplexEmpty(chunk);
	//hipFloatComplex /*kddata,*kddout,*/ *krefsig =  getComplexEmpty(2*chunk);
	//repMat(getReferenceSignal("data/rref3sig2.dat","data/iref3sig2.dat",100,ftpoint),khout,ftpoint,chunk,1);
	repMat(hamming(ftpoint),khout,ftpoint,chunk,1);
	//zepadMat(khout,krefsig,1300,ftpoint,2);
	gettimeofday(&t2, 0);
	double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("\n\n %f \n\n",time);
	//
	// TODO: Allocate and Init ddata n ddout
	//hipMalloc((void**)&kddata,ftpoint*sizeof(hipFloatComplex));
	//hipMalloc((void**)&kddout,chunk*sizeof(hipFloatComplex));
	//hipMemcpy(krefsig,kddata,ftpoint*sizeof(hipFloatComplex),hipMemcpyHostToDevice);
	//
	//repmat<<<dimGrid,dimBlock>>>(kddata,kddout,ftpoint,chunk);
	//
	//hipMemcpy(khout,kddata,chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost);
	//
	//
	printf("\n\n[");
	for(int i=0; i<2*ftpoint;i++)
	{
		printf("%f  ",hipCrealf(khout[i]));
		if((i>0) && (i%ftpoint==0))printf("]\n\n[");
	}
	printf("]\n\n");
	//
	//hipFree(kddata);
	//hipFree(kddout);
	free(khout);
	//free(krefsig);
	//hipDeviceReset();
}
//
//%%%%%%%%% Power Spectrum Stream Processing using CUDA GPUs %%%%%%%%%%%%%
//
double _JSSHamming(vector<hipFloatComplex*> hdata, int dsize,int chunk,int hmsize, int padding)
{
	//
	struct timeval t1, t2;
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        //
	int grid1 = ceil(padding*chunk/(float)threadsPerBlock);
	dimGrid.x = grid1;
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //
	//hdata.resize(dsize); // This must be done outside with data initialization/generation
	//hout.resize(dsize); // Might be a good idea to initialize outside of this function...
	hipStream_t optisdr_streams[dsize];	
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//drout.resize(dsize);
	//
	hipFloatComplex *dhin;	
	//
	hipFloatComplex *khout =  getComplexEmpty(padding*chunk);
	repMat(hamming(hmsize),khout,hmsize,chunk,padding); // Create Hamming and repeat up to chunk size...
	//
	hipMalloc((void**)&dhin,padding*chunk*sizeof(hipFloatComplex));
	hipMemcpy(dhin,khout,padding*chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice);
	//
	// Trying to Create Page-Locked std::vector - No need for this if we need simple malloc()...
	//
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],padding*chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);//hipHostAlloc(...);
		//hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipMalloc((void**)&ddata[i],padding*chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],padding*chunk*sizeof(hipFloatComplex));
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Execution
	//
	for(int i = 0; i<dsize; i++)
	{
		hipMemcpyAsync(ddata[i],hdata[i],padding*chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	// Execution
	//	
	gettimeofday(&t1, 0);
	//
	for(int i = 0; i<dsize; i++)
	{		
		//
		//
		complexvector_multiply1d1d<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dhin,ddata[i],dout[i],padding*chunk);
		//
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	gettimeofday(&t2, 0);
	double GPUTime = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec); ///1000000.0;
	//
	for(int i = 0; i<dsize; i++)
	{		
		//
		hipMemcpyAsync(hdata[i],dout[i],padding*chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		//hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		//
		hipFree(ddata[i]);
		hipFree(dout[i]);
		
		//
	}
	hipFree(dhin);
	free(khout);
	return GPUTime;
}
//
//
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%% Hanning Window using CUDA %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%% Blackman Window using CUDA %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%% Power Spectrum Stream Processing using CUDA GPUs %%%%%%%%%%%%%
//
double _10logAbs(vector<hipFloatComplex*> hdata, int dsize,int chunk,hipFloatComplex Vpp,hipFloatComplex divV, float logV)
{
	//
	struct timeval t1, t2;
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        //
	int grid1 = ceil(chunk/(float)threadsPerBlock);
	dimGrid.x = grid1;
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //
	//hdata.resize(dsize); // This must be done outside with data initialization/generation
	//hout.resize(dsize); // Might be a good idea to initialize outside of this function...
	hipStream_t optisdr_streams[dsize];	
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	// Trying to Create Page-Locked std::vector - No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);//hipHostAlloc(...);
		//hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Execution
	//
	for(int i = 0; i<dsize; i++)
	{
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	// Execution
	//	
	gettimeofday(&t1, 0);
	for(int i = 0; i<dsize; i++)
	{		
		//
		//
		_20log10fabs<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(ddata[i],dout[i],chunk,Vpp,divV,logV);
		//
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	gettimeofday(&t2, 0);
	double GPUTime = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec); ///1000000.0;
	//
	for(int i = 0; i<dsize; i++)
	{		
		//
		hipMemcpyAsync(hdata[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		//hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		//
		hipFree(ddata[i]);
		hipFree(dout[i]);
		//
	}
	return GPUTime;
}
//
//
double _10logAbs(vector<hipFloatComplex*> hdata, int dsize,int chunk)
{
	//
	struct timeval t1, t2;
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        //
	int grid1 = ceil(chunk/(float)threadsPerBlock);
	dimGrid.x = grid1;
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //
	//hdata.resize(dsize); // This must be done outside with data initialization/generation
	//hout.resize(dsize); // Might be a good idea to initialize outside of this function...
	hipStream_t optisdr_streams[dsize];	
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	// Trying to Create Page-Locked std::vector - No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);//hipHostAlloc(...);
		//hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Execution
	//
	for(int i = 0; i<dsize; i++)
	{
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	// Execution
	//	
	gettimeofday(&t1, 0);
	for(int i = 0; i<dsize; i++)
	{		
		//
		//
		_10logabs<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(ddata[i],dout[i],chunk);
		//
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	gettimeofday(&t2, 0);
	double GPUTime = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec); ///1000000.0;
	//
	for(int i = 0; i<dsize; i++)
	{		
		//
		hipMemcpyAsync(hdata[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		//hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		//
		hipFree(ddata[i]);
		hipFree(dout[i]);
		//
	}
	return GPUTime;
}
//
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%% FFT Stream Processing using CUDA GPUs %%%%%%%%%%%%%%%%%%%%%%%%
//
void streamprocessor(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> hout, int dsize,int chunk)
{
	//
	struct timeval t1, t2;
	gettimeofday(&t1, 0);
	//hdata.resize(dsize); // This must be done outside with data initialization/generation
	//hout.resize(dsize); // Might be a good idea to initialize outside of this function...
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);	
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	// Trying to Create Page-Locked std::vector - No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);//hipHostAlloc(...);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	
	for(int i = 0; i<dsize; i++)
	{
		hipfftPlan1d(&plans[i],chunk,HIPFFT_C2C,1);
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
    		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		//free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	gettimeofday(&t2, 0);
	double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	double time2 = ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000000.0;
	printf("\n[SDR_DSL_INFO]$ Exec. Time for StreamProcessed FFT = %f s ~= %f...!\n", time,time2);
	//hipDeviceReset();
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//%%%%%%%%%%%%%%%%% Batched FFT Stream processing using CUDA Streams and Pinned-Memory %%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
double BatchedFFT(vector<hipFloatComplex*> hdata,int dsize,int chunk, int ftpoint)
{
	//
	struct timeval t1, t2;
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	int N_SIGS = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//hipfftPlan1d(&plans[i],chunk,HIPFFT_C2C,1);
		//hipfftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,HIPFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C, N_SIGS);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	// Execution
	//	
	gettimeofday(&t1, 0);
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	gettimeofday(&t2, 0);
	double GPUTime = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec); ///1000000.0;
	//
	for(int i = 0; i<dsize; i++)
	{		
		//
		hipMemcpyAsync(hdata[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		//
	}
	//
	//
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		//hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		//free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	return GPUTime;
}
//
// Generic Stream for FFT
//
void streamprocessor(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	//struct timeval t1, t2;
	//gettimeofday(&t1, 0);
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	int N_SIGS = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	vector<hipFloatComplex*> ddata,dout;
	ddata.resize(dsize);
	dout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//hipfftPlan1d(&plans[i],chunk,HIPFFT_C2C,1);
		//hipfftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,HIPFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C, N_SIGS);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//hipStreamCreate(&optisdr_streams[i]);
		//hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		//free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	//gettimeofday(&t2, 0);
	//double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	//double time2 = ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000000.0;
	//printf("\n[SDR_DSL_INFO]$ Exec. Time for StreamProcessed FFT = %f s ~= %f...!\n", time,time2);
	//hipDeviceReset();
}
//
//// XCorr Conjugate and Multiply
__global__ void xcorrmultiply(hipFloatComplex *_cx,hipFloatComplex *_cy,hipFloatComplex *outp,int N)
{
	//
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	//
	if(index < N)
	{
		outp[index] = hipCmulf(_cx[index],hipConjf(_cy[index]));
	}
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
double XCorrSP(vector<hipFloatComplex*> hdata, hipFloatComplex* refsig, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	struct timeval kernt1, kernt2;
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	//printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	hipFloatComplex *drefsig,*dftrefsig;
	hipMalloc((void**)&drefsig,chunk*sizeof(hipFloatComplex));
	//hipMalloc((void**)&dftrefsig,chunk*sizeof(hipFloatComplex));
	//hipHostRegister(refsig,chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
	hipMemcpy(drefsig,refsig,chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice);
	//hipMalloc((void**)&drout,chunk*sizeof(hipFloatComplex));
	//
	vector<hipFloatComplex*> ddata,dout,drout;
	ddata.resize(dsize);
	dout.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//hipfftPlan1d(&plans[i],chunk,HIPFFT_C2C,1);
		//hipfftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,HIPFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//hipStreamCreate(&optisdr_streams[i]);
		//hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&drout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	//
	gettimeofday(&kernt1, 0);
	//res = hipfftExecC2C(plans[0],drefsig,dftrefsig,HIPFFT_FORWARD);
	//if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		//complexvector_multiply1d1d<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],dftrefsig,drout[i],chunk);
		xcorrmultiply<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],drefsig,drout[i],chunk);
		//
		res = hipfftExecC2C(plans[i],drout[i],ddata[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Inverse transform fail.\n");}
		//
		// TODO: Try using the Block Size =128, i.e optisdrifftscale<<<chunk/128,128,0,optisdr_treams[i]>>>(...);
        	optisdrifftscale<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(ddata[i],dout[i],ftpoint,chunk);
		//
		//hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	gettimeofday(&kernt2, 0);
	double GPUTime = (1000000.0*(kernt2.tv_sec-kernt1.tv_sec) + kernt2.tv_usec-kernt1.tv_usec); ///1000000.0;
	for(int i = 0; i<dsize; i++)
	{		
		//
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		hipFree(drout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	hipHostUnregister(refsig);
	hipFree(drefsig);
	return GPUTime;
	//gettimeofday(&t2, 0);
	//double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	//double time2 = ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000000.0;
	//printf("\n[SDR_DSL_INFO]$ Exec. Time for StreamProcessed FFT = %f s ~= %f...!\n", time,time2);
	//hipDeviceReset();
}
//
//
double XCorrSP2(vector<hipFloatComplex*> hdata, hipFloatComplex* refsig, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	struct timeval kernt1, kernt2;
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	//printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	//
	hipFloatComplex *drefsig,*dftrefsig;
	hipMalloc((void**)&drefsig,chunk*sizeof(hipFloatComplex));
	hipMalloc((void**)&dftrefsig,chunk*sizeof(hipFloatComplex));
	//hipHostRegister(refsig,chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
	hipMemcpy(drefsig,refsig,chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice);
	//hipMalloc((void**)&drout,chunk*sizeof(hipFloatComplex));
	//
	vector<hipFloatComplex*> ddata,dout,drout;
	ddata.resize(dsize);
	dout.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//hipfftPlan1d(&plans[i],chunk,HIPFFT_C2C,1);
		//hipfftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,HIPFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1, ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,chunk/ftpoint);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//hipStreamCreate(&optisdr_streams[i]);
		//hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&drout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	//
	gettimeofday(&kernt1, 0);
	res = hipfftExecC2C(plans[0],drefsig,dftrefsig,HIPFFT_FORWARD);
	if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		//complexvector_multiply1d1d<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],dftrefsig,drout[i],chunk);
		xcorrmultiply<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],dftrefsig,drout[i],chunk);
		//
		res = hipfftExecC2C(plans[i],drout[i],ddata[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Inverse transform fail.\n");}
		//
		// TODO: Try using the Block Size =128, i.e optisdrifftscale<<<chunk/128,128,0,optisdr_treams[i]>>>(...);
        	optisdrifftscale<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(ddata[i],dout[i],ftpoint,chunk);
		//
		//hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	gettimeofday(&kernt2, 0);
	double GPUTime = (1000000.0*(kernt2.tv_sec-kernt1.tv_sec) + kernt2.tv_usec-kernt1.tv_usec); ///1000000.0;
	for(int i = 0; i<dsize; i++)
	{		
		//
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		hipFree(drout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	hipHostUnregister(refsig);
	hipFree(drefsig);
	return GPUTime;
	//gettimeofday(&t2, 0);
	//double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
	//double time2 = ((t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec))/1000000.0;
	//printf("\n[SDR_DSL_INFO]$ Exec. Time for StreamProcessed FFT = %f s ~= %f...!\n", time,time2);
	//hipDeviceReset();
}
//
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// Creating a Dummy h vector for computing the Hilbert Transform
//
hipFloatComplex* getHilbertHVector(int _dlen)
{
	//
	hipFloatComplex *h = (hipFloatComplex *)malloc(_dlen*sizeof(hipFloatComplex));
	//
	h[0] = make_hipFloatComplex(1.0f,0.0f);
	h[_dlen/2] = make_hipFloatComplex(1.0f,0.0f);
	//
	int i = 1;
	int j = (_dlen/2)+1;
	//
	while(i<(_dlen/2))
	{
		h[i] = make_hipFloatComplex(2.0f,0.0f);
		i = i + 1;
		h[j] = make_hipFloatComplex(0.0f,0.0f);
		j = j + 1;
	}
	return h;
}
//
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
double HilbertSP(vector<hipFloatComplex*> hdata, vector<hipFloatComplex*> hout, int dsize,int chunk, int ftpoint)
{
	//
	struct timeval kernt1, kernt2;
	dim3 dimBlock, dimGrid;
    	int threadsPerBlock;//, blocksPerGrid;
	//
	dimBlock.x = 1024;
	//dimBlock.y = 1024; // Uncomment for 2-D Grid
	threadsPerBlock = dimBlock.x*dimBlock.y*dimBlock.z;
        // Set up grid
        // Here you will have to set up the variables: dimGrid and blocksPerGrid
	int grid1 = ceil(chunk/(float)threadsPerBlock); // For 1-D Grid
	//int grid1 = ceil(sqrt(numEls/(float)threadsPerBlock)); // For 2-D Grid
	//int grid1 = ceil(cbrt(numEls/(float)threadsPerBlock)); // For 3-D Grid
	dimGrid.x = grid1;
	//dimGrid.y = grid1; // uncomment for 2-D grid
	//dimGrid.z = grid1; // uncomment for 3-D grid
	//
	//blocksPerGrid = dimGrid.x*dimGrid.y*dimGrid.z; // Never really use this
        //TODOMSG("Calculate grid dimensions")
	//printf("\nThe Grid Dim: [%d]\nThreadsPerBlock: [%d]\nBlocksPerGrid: [%d]\n\n",grid1,threadsPerBlock,blocksPerGrid);
	//
	//	
	hipFloatComplex* h = resizeVector(getHilbertHVector(ftpoint),ftpoint,chunk);
	//
	hipStream_t optisdr_streams[dsize];
	hipfftHandle* plans = (hipfftHandle*) malloc(sizeof(hipfftHandle)*dsize);
	//int N_SIGS2 = chunk/ftpoint; // Chunk must be a multiple of ftpoint
	int n[1] = {ftpoint};
	int sigLen = chunk/ftpoint; // Make sure chunk is multiple of 2, better done at DSL level
	//
	hipFloatComplex *dh;
	hipMalloc((void**)&dh,chunk*sizeof(hipFloatComplex));
	hipHostRegister(h,chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
	//hipMalloc((void**)&drout,chunk*sizeof(hipFloatComplex));
	//
	vector<hipFloatComplex*> ddata,dout,dhout,drout;
	ddata.resize(dsize);
	dout.resize(dsize);
	dhout.resize(dsize);
	drout.resize(dsize);
	//
	for(int i = 0;i<dsize;i++)
	{
		hipStreamCreate(&optisdr_streams[i]);
	}
	//
	// Creating cuFFT plans and sets them in streams
	//
	hipfftResult res;	
	for(int i = 0; i<dsize; i++)
	{
		//hipfftPlan1d(&plans[i],chunk,HIPFFT_C2C,1);
		//hipfftPlanMany(&plans[i],1,n,NULL,1,SIG_LEN,NULL,1,SIG_LEN,HIPFFT_C2C,N_SIGS);
		res = hipfftPlanMany(&plans[i], 1, n,
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		NULL, 1,ftpoint, //advanced data layout, NULL shuts it off
    		HIPFFT_C2C,sigLen);
   		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Plan create fail.\n");}
		//
		hipfftSetStream(plans[i],optisdr_streams[i]);
	}
	// Trying to Create Page-Locked std::vector
	// No need for this if we need simple malloc()...
	for(int i = 0;i<dsize;i++)
	{
		// TODO: Check Out hipHostMalloc(...), cudaMallocManage(...), ...
		hipHostRegister(hdata[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable); // Page-Locked Mem.
		hipHostRegister(hout[i],chunk*sizeof(hipFloatComplex),hipHostRegisterPortable);
		//hipStreamCreate(&optisdr_streams[i]);
		//hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
	}
	//
	hipMemcpyAsync(dh,h,chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[0]);
	for(int i = 0;i<dsize;i++)
	{
		hipMalloc((void**)&ddata[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&dhout[i],chunk*sizeof(hipFloatComplex));
		hipMalloc((void**)&drout[i],chunk*sizeof(hipFloatComplex));
		hipMemcpyAsync(ddata[i],hdata[i],chunk*sizeof(hipFloatComplex),hipMemcpyHostToDevice,optisdr_streams[i]);
		//
	}
	//
	gettimeofday(&kernt1, 0);
	// Execution
	for(int i = 0; i<dsize; i++)
	{		
		//
		res = hipfftExecC2C(plans[i],ddata[i],dout[i],HIPFFT_FORWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		complexvector_multiply1d1d<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(dout[i],dh,dhout[i],chunk);
		//
		res = hipfftExecC2C(plans[i],dhout[i],drout[i],HIPFFT_BACKWARD);
		if (res != HIPFFT_SUCCESS) {printf("[SDR_DSL_INFO]$ Forward transform fail.\n");}
		//
		optisdrifftscale<<<dimGrid,dimBlock,0,optisdr_streams[i]>>>(drout[i],dout[i],ftpoint,chunk);
		//
		//hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	//
	gettimeofday(&kernt2, 0);
	double GPUTime = (1000000.0*(kernt2.tv_sec-kernt1.tv_sec) + kernt2.tv_usec-kernt1.tv_usec); ///1000000.0;
	//
	for(int i = 0; i<dsize; i++)
	{		
		//
		hipMemcpyAsync(hout[i],dout[i],chunk*sizeof(hipFloatComplex),hipMemcpyDeviceToHost,optisdr_streams[i]);
		hipStreamSynchronize(optisdr_streams[i]);
	}
	// Releasing Computing Resources
	for(int i = 0; i < dsize; i++)
	{
		hipStreamDestroy(optisdr_streams[i]);
		hipHostUnregister(hdata[i]);
		hipHostUnregister(hout[i]);
		hipFree(ddata[i]);
		hipFree(dout[i]);
		hipFree(dhout[i]);
		hipFree(drout[i]);
		free(hdata[i]);
		hipfftDestroy(plans[i]);
	}
	hipHostUnregister(h);
	hipFree(dh);
	free(h);
	//
	return GPUTime;
}
//
hipError_t cuda_main()
{
    // generate 16M random numbers on the host
    thrust::host_vector<int> h_vec(1 << 24);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);
  
    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec;
  
    // sort data on the device (805 Mkeys/sec on GeForce GTX 480)
    thrust::sort(d_vec.begin(), d_vec.end());
  
    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
  
    return hipGetLastError();
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
#endif
