#include "hip/hip_runtime.h"
#ifndef _DELITE_CUDA_
#define _DELITE_CUDA_

#include "DeliteCuda.h"

using namespace std;

list<void*>* lastAlloc = new list<void*>();
queue<FreeItem>* freeList = new queue<FreeItem>();
map<void*,list<void*>*>* cudaMemoryMap = new map<void*,list<void*>*>();

void addEvent(hipStream_t fromStream, hipStream_t toStream) {
  hipEvent_t event;
  hipEventCreateWithFlags(&event, hipEventDisableTiming);
  hipEventRecord(event, fromStream);
  hipStreamWaitEvent(toStream, event, 0);
  hipEventDestroy(event);
}

hipEvent_t addHostEvent(hipStream_t stream) {
  hipEvent_t event;
  hipEventCreateWithFlags(&event, hipEventDisableTiming | hipEventBlockingSync);
  hipEventRecord(event, stream);
  return event;
}

void freeCudaMemory(FreeItem item) {
  list< pair<void*,bool> >::iterator iter;
  for (iter = item.keys->begin(); iter != item.keys->end(); iter++) {
    //cout << "object ref: " << (long) *iter << endl;
    if(cudaMemoryMap->find((*iter).first) != cudaMemoryMap->end()) {
      list<void*>* freePtrList = cudaMemoryMap->find((*iter).first)->second;
      list<void*>::iterator iter2;
      for (iter2 = freePtrList->begin(); iter2 != freePtrList->end(); iter2++) {
        void* freePtr = *iter2;
        hipFree(freePtr);
        //if (hipFree(freePtr) != hipSuccess)
        //    cout << "bad free pointer: " << (long) freePtr << endl;
        //else
        //cout << "freed successfully: " << (long) freePtr << endl;
      }
      cudaMemoryMap->erase((*iter).first);
      delete freePtrList;
      if(!((*iter).second)) free((*iter).first);
    }
  }
  delete item.keys;
}

// collects cuda memory allocated for kernels completed at the moment
void DeliteCudaGC(void) {
  while (freeList->size() != 0) {
    FreeItem item = freeList->front();
    if (hipEventQuery(item.event) != hipSuccess) {
      break;
    }
    freeList->pop();
    hipEventDestroy(item.event);
    freeCudaMemory(item);
  }
}

// collects all garbages and checks no remaining allocations left
void DeliteCudaCheckGC(void) {
  DeliteCudaGC();
  if(freeList->size() != 0) 
    cout << "WARNING: memory not collectd : count " << freeList->size() << endl; 
}

// allocates a chunk of cuda device memory
// run GC before allocation
void DeliteCudaMalloc(void** ptr, size_t size) {

  DeliteCudaGC();

  while (hipMalloc(ptr, size) != hipSuccess) {
    if (freeList->size() == 0) {
      cout << "FATAL: Insufficient device memory" << endl;
      exit(-1);
    }
    FreeItem item = freeList->front();
    freeList->pop();

    while (hipEventQuery(item.event) != hipSuccess) {
      hipEventSynchronize(item.event);
    }
    hipEventDestroy(item.event);
    freeCudaMemory(item);
  }
  lastAlloc->push_back(*ptr);
}

size_t cudaHeapSize = 1024*1204;

/* Implementations for temporary memory management */
#define CUDAMEM_ALIGNMENT 64
char *tempCudaMemPtr;
size_t tempCudaMemOffset;
size_t tempCudaMemSize;

// initialize cuda temporary device memory
void tempCudaMemInit(double tempMemRate) {
  DeliteCudaProfInit();
  size_t free, total;
  hipMemGetInfo(&free, &total);
  tempCudaMemSize = total * tempMemRate;
  //cout << "initializing cuda temp mem.." << endl;
  //cout << "Free:" << free << endl;
  //cout << "Total:" << total << endl;
  //cout << "tempMemSize:" << tempCudaMemSize << endl;
  tempCudaMemOffset = 0;
  if(hipMalloc(&tempCudaMemPtr, tempCudaMemSize) != hipSuccess) {
    cout << "FATAL (tempCudaMemInit): Insufficient device memory for tempCudaMem" << endl;
    exit(-1);
  }
  //cout << "finished temp init" << endl;
}

// free cuda temporary memory
void tempCudaMemFree(void) {
  if(hipFree(tempCudaMemPtr) != hipSuccess) {
    cout << "FATAL (tempCudaMemFree): Failed to free temporary memory" << endl;
    exit(-1);
  }
}

// reset cuda temporary memory (called by each multiloop)
void tempCudaMemReset(void) {
  tempCudaMemOffset = 0;
}

// return the size of available temporary memory
size_t tempCudaMemAvailable(void) {
  return (tempCudaMemSize - tempCudaMemOffset - CUDAMEM_ALIGNMENT);
}

// allocates cuda device memory from temporary space
void DeliteCudaMallocTemp(void** ptr, size_t size) {
  size_t alignedSize = CUDAMEM_ALIGNMENT * (1 + size / CUDAMEM_ALIGNMENT);
  if(tempCudaMemOffset + alignedSize > tempCudaMemSize) {
    cout << "FATAL(DeliteCudaMallocTemp): Insufficient device memory for tempCudaMem" << endl;
    exit(-1);
  }
  else {
    *ptr = tempCudaMemPtr + tempCudaMemOffset;
    tempCudaMemOffset += alignedSize;
  }
}

// variables for cuda host memory
char* bufferStart = 0;
char* bufferEnd;
char* bufferCurrent;

// initialize cuda host memory (page-mapped system memory for asynchronous copy)
void hostInit() {
  size_t free, total;
  hipMemGetInfo(&free, &total);
  // allocate the host memory as much as the device memory (make it parameter?)
  hipHostAlloc(&bufferStart, total, hipHostMallocDefault);
  bufferEnd = bufferStart + total;
  bufferCurrent = bufferStart;
}

// free cuda host memory
void cudaHostMemFree(void) {
  hipHostFree(bufferStart);
  bufferStart = NULL;
}

// allocate cuda host memory
void DeliteCudaMallocHost(void** ptr, size_t size) {
  size_t alignedSize = CUDAMEM_ALIGNMENT * (1 + size / CUDAMEM_ALIGNMENT);
  if (bufferStart == 0) hostInit();
  if ((bufferCurrent + alignedSize) > bufferEnd)
    bufferCurrent = bufferStart;
  *ptr = bufferCurrent;
  bufferCurrent += alignedSize;
}

void DeliteCudaMemcpyHtoDAsync(void* dptr, void* sptr, size_t size) {
  hipMemcpyAsync(dptr, sptr, size, hipMemcpyHostToDevice, h2dStream);
}

void DeliteCudaMemcpyDtoHAsync(void* dptr, void* sptr, size_t size) {
  hipMemcpyAsync(dptr, sptr, size, hipMemcpyDeviceToHost, d2hStream);
  hipStreamSynchronize(d2hStream);
}

void DeliteCudaMemcpyDtoDAsync(void *dptr, void* sptr, size_t size) {
  hipMemcpyAsync(dptr, sptr, size, hipMemcpyDeviceToDevice, kernelStream);
}

void DeliteCudaMemset(void *ptr, int value, size_t count) {
  hipMemset(ptr,value,count);
}

void DeliteCudaCheckError(void) {
  hipDeviceSynchronize();
  if (hipPeekAtLastError() != hipSuccess) {
    cout << "DeliteCuda execution failed: " << hipGetErrorString(hipPeekAtLastError()) << endl;
    exit(-1);
  }
}

struct timeval start_t;
struct timeval end_t;
void DeliteCudaTic(void) {
  hipDeviceSynchronize();
  gettimeofday(&start_t,NULL);
}

void DeliteCudaToc(void) {
  hipDeviceSynchronize();
  gettimeofday(&end_t,NULL);
  double exetime = (end_t.tv_sec*1000000+end_t.tv_usec) - (start_t.tv_sec*1000000+start_t.tv_usec);
  cout << "DeliteCudaTimer(static) " << " " << (exetime)/1000.0 << " ms" << endl; 
}

//map<string,double> *cudaTimerMap = new map<string,double>();

char **ticName;
double *ticStart;
int ticIdx;

void DeliteCudaProfInit(void) {
  ticName = new char*[1024];
  ticStart = new double[1024];
  ticIdx = 0;
}

void DeliteCudaTic(char *name) {
  struct timeval t;

  hipDeviceSynchronize();
  gettimeofday(&t,NULL);
  ticStart[ticIdx] = t.tv_sec*1000000+t.tv_usec; 
  ticName[ticIdx] = name;
  ticIdx += 1;
  //cudaTimerMap->insert(pair<string,double>(name, t.tv_sec*1000000+t.tv_usec));
}

void DeliteCudaToc(char *name) {
  struct timeval t;
  //map<string,double>::iterator it = cudaTimerMap->find(name);
  //double start = (it==cudaTimerMap->end()) ? 0 : it->second;
  //if(start != 0) cudaTimerMap->erase(it);
  hipDeviceSynchronize();
  gettimeofday(&t,NULL);
  double end = t.tv_sec*1000000+t.tv_usec;
  ticIdx -= 1;
  cout << "DeliteCudaTimer " <<ticName[ticIdx]<< " : " << (end-ticStart[ticIdx])/1000.0 << " ms" << endl; 
}
//

// TODO: Remove this kernel from here by generate it 
__global__ void kernel_offset(int *key, int *idx, int *offset, int size) {

  int idxX = threadIdx.x + blockIdx.x*blockDim.x;

  if(idxX == 0) {
    offset[1] = 0;
  }
  else if(idxX < size) {
    int keyVal = key[idxX];
    int keyValPrev = key[idxX-1]; 
    if(keyVal != keyValPrev) {
      offset[keyVal+1] = idxX;
    }
  }
  if(idxX == size-1) {
    int keyVal = key[idxX];
    offset[0] = keyVal+1;
    offset[keyVal+2] = size;
  }
}

#endif