#include <jni.h>
#include <hip/hip_runtime.h>

extern "C" JNIEXPORT void JNICALL Java_ppl_delite_runtime_executor_AccExecutionThread_initializeDevice(JNIEnv* env, jobject obj, jint deviceNum);

JNIEXPORT void JNICALL Java_ppl_delite_runtime_executor_AccExecutionThread_initializeDevice(JNIEnv* env, jobject obj, jint deviceNum) {
	//chose device num
  if(hipSuccess != hipSetDevice(deviceNum)) {
    printf("FATAL : GPU device could not be initialized. \n");	
    exit(1);
  }
	//reset the device
  if(hipSuccess != hipDeviceReset()) {
    printf("FATAL : hipDeviceReset() failed \n");	
    exit(1);
  }
  //set device options
  if(hipSuccess != hipSetDeviceFlags(hipDeviceScheduleBlockingSync)) {
    printf("FATAL : GPU device has crashed (cudaSetDviceFlags). \n");	
    exit(1);
  }
  //called to initialize the device (can take a while)
  if(hipSuccess != hipDeviceSynchronize()) {
    printf("FATAL : GPU device has crashed (hipDeviceSynchronize). \n");	
    exit(1);
  }
}
