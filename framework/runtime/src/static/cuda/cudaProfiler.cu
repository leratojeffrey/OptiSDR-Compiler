#ifndef DELITE_CUDA_PROFILER_INCLUDED
#define DELITE_CUDA_PROFILER_INCLUDED

#include <sys/time.h>

struct timeval myprofiler_start, myprofiler_end;
void printTime(void) {
  printf("Time : %ld [us]\n", ((myprofiler_end.tv_sec * 1000000 + myprofiler_end.tv_usec) - (myprofiler_start.tv_sec * 1000000 + myprofiler_start.tv_usec)));
}
void printTime(char *str) {
  printf("%s : Time : %ld [us]\n", str, ((myprofiler_end.tv_sec * 1000000 + myprofiler_end.tv_usec) - (myprofiler_start.tv_sec * 1000000 + myprofiler_start.tv_usec)));
}
/*
void printTime(char *str) {
  sprintf(str, "%s : Time : %ld [us]\n", str, ((myprofiler_end.tv_sec * 1000000 + myprofiler_end.tv_usec) - (myprofiler_start.tv_sec * 1000000 + myprofiler_start.tv_usec)));
}
*/
void mytic(void) {
  hipDeviceSynchronize();
  gettimeofday(&myprofiler_start,NULL);
}

void mytoc(void) {
  hipDeviceSynchronize();
  gettimeofday(&myprofiler_end,NULL);
  printTime();
}

void mytoc(char *str) {
  hipDeviceSynchronize();
  gettimeofday(&myprofiler_end,NULL);
  printTime(str);
}

#endif
